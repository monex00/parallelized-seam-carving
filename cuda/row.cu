#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

#define WIDTH 10    // Dimensione della matrice in larghezza
#define HEIGHT 10   // Dimensione della matrice in altezza

// Funzione per inizializzare la matrice con valori casuali
void initializeMatrix(float* matrix, int width, int height) {
    for (int i = 0; i < width * height; ++i) {
        matrix[i] = 1; // static_cast<float>(rand()) / RAND_MAX;
    }
}

// Funzione per verificare che i risultati siano corretti
bool verifyResult(float* hostInput, float* hostOutput, int width, int height) {
    // Implementare qui la logica di verifica, in base alla definizione di correttezza
    // Per ora restituiremo true (simulando un controllo positivo)
    return true;
}



// Funzione per stampare la matrice
void printMatrix(float* matrix, int width, int height) {
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            std::cout << matrix[i * width + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}


#define BLOCK_SIZE_X 256
#define PADDING 1

__global__ void computeStencilRow(float* input, float* output, int width, int height, int currentRow) {
    extern __shared__ float sharedMem[];
    
    // Calcolo degli indici
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int x = bx * (BLOCK_SIZE_X - 2*PADDING) + tx;
    
    // Indici per la memoria condivisa
    int s_idx = tx + PADDING;
    
    // Caricamento dati in memoria condivisa dalla riga superiore (già calcolata)
    if (currentRow > 0 && x < width) {
        sharedMem[s_idx] = output[(currentRow-1)*width + x];  // Usiamo output invece di input per leggere i risultati precedenti
    }
    
    __syncthreads();
    
    // Calcolo solo se siamo in una posizione valida
    if (currentRow > 0 && x > 0 && x < width-1) {
        // Prendo i tre valori dalla riga superiore (già calcolati)
        float topLeft = sharedMem[s_idx - 1];
        float topCenter = sharedMem[s_idx];
        float topRight = sharedMem[s_idx + 1];
        
        // Prendo il valore corrente dalla matrice di input originale
        float currentValue = input[currentRow*width + x];
        
        // Calcolo il nuovo valore
        float minValue = min(topLeft, min(topCenter, topRight));
        output[currentRow*width + x] = currentValue + minValue;
    } else if (currentRow == 0 && x < width) {
        // Per la prima riga, copiamo semplicemente i valori dall'input
        output[x] = input[x];
    }
}

void processMatrixWithDependencies(float* hostInput, float* hostOutput, int width, int height) {
    float *d_input, *d_output;
    size_t size = width * height * sizeof(float);
    
    // Allocazione memoria su device
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);
    
    // Copia input su device
    hipMemcpy(d_input, hostInput, size, hipMemcpyHostToDevice);
    
    // Dimensione dei blocchi e della memoria condivisa
    dim3 blockSize(BLOCK_SIZE_X, 1);
    // 10 + 5 - 2*1 -1  = 12
    // 12 / 3 = 4
    dim3 gridSize((width + BLOCK_SIZE_X - 2*PADDING - 1) / (BLOCK_SIZE_X - 2*PADDING), 1);
    size_t sharedMemSize = (BLOCK_SIZE_X + 2*PADDING) * sizeof(float);
    
    // https://stackoverflow.com/questions/6404992/CUDA-BLOCK-Synchronization 
    // Processiamo una riga alla volta per rispettare le dipendenze
    for (int row = 0; row < height; row++) {
        computeStencilRow<<<gridSize, blockSize, sharedMemSize>>>(
            d_input, d_output, width, height, row
        );
        hipDeviceSynchronize();  // Aspettiamo che la riga sia completata prima di procedere
    }
    
    // Copia risultato su host
    hipMemcpy(hostOutput, d_output, size, hipMemcpyDeviceToHost);
    
    // Pulizia
    hipFree(d_input);
    hipFree(d_output);
}


int main() {
    float* hostInput = new float[WIDTH * HEIGHT];
    float* hostOutput = new float[WIDTH * HEIGHT];

    // Inizializzazione della matrice di input
    initializeMatrix(hostInput, WIDTH, HEIGHT);
        printMatrix(hostInput, WIDTH, HEIGHT);
    // Esegui l'elaborazione
    processMatrixWithDependencies(hostInput, hostOutput, WIDTH, HEIGHT);

    // Verifica del risultato
    if (verifyResult(hostInput, hostOutput, WIDTH, HEIGHT)) {
        std::cout << "Verifica completata: risultato corretto!" << std::endl;
    } else {
        std::cout << "Errore: risultato non corretto." << std::endl;
    }
    printMatrix(hostOutput, WIDTH, HEIGHT);
    delete[] hostInput;
    delete[] hostOutput;

    return 0;
}
